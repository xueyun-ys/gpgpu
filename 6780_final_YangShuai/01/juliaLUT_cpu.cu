/*references:
 * NVIDIA Corporation code: Julia.
 Letures and notes from 6780 GPGPU & 6040 CGI
 libraries: common files from class
 *
 */

 #include <time.h>
 #include <sys/time.h>
 #include <stdlib.h>
 #include <iostream>
 #include ""
 #include <memory>
 #include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <values.h>

#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000

struct LUT {
    float colori=0.0;

    LUT( float a ) : colori(a)  {}
    float red( void )
    {
      	if (colori/2.0 <0.1)
      	    return 0.4;
      	else if (colori/2.0 <0.2)
      	    return 0.1;
      	else if (colori/2.0 <0.3)
      	    return 0.2;
      	else if (colori/2.0 <0.4)
      	    return 0.4;
      	else if (colori/2.0 <0.5)
      	    return 0.7;
      	else if (colori/2.0 <0.6)
      	    return 0.9;
      	else if (colori/2.0 <0.7)
      	    return 0.65;
      	else if (colori/2.0 <0.8)
      	    return 0.47;
      	else if (colori/2.0 <0.9)
      	    return 0.78;
      	else if (colori/2.0 <= 1.0)
      	    return 0.34;
    }
    float green( void )
    {
      	if (colori/2.0 <0.1)
      	    return 0.6;
      	else if (colori/2.0 <0.2)
      	    return 0.1;
      	else if (colori/2.0 <0.3)
      	    return 0.46;
      	else if (colori/2.0 <0.4)
      	    return 0.38;
      	else if (colori/2.0 <0.5)
      	    return 0.37;
      	else if (colori/2.0 <0.6)
      	    return 0.6;
      	else if (colori/2.0 <0.7)
      	    return 0.65;
      	else if (colori/2.0 <0.8)
      	    return 0.91;
      	else if (colori/2.0 <0.9)
      	    return 0.34;
      	else if (colori/2.0 <= 1.0)
      	    return 0.56;

    }
    float blue( void )
    {
      	if (colori/2.0 <0.1)
      	    return 0.8;
      	else if (colori/2.0 <0.2)
      	    return 0.23;
      	else if (colori/2.0 <0.3)
      	    return 0.2;
      	else if (colori/2.0 <0.4)
      	    return 0.62;
      	else if (colori/2.0 <0.5)
      	    return 0.32;
      	else if (colori/2.0 <0.6)
      	    return 0.51;
      	else if (colori/2.0 <0.7)
      	    return 0.13;
      	else if (colori/2.0 <0.8)
      	    return 0.93;
      	else if (colori/2.0 <0.9)
      	    return 0.46;
      	else if (colori/2.0 <= 1.0)
      	    return 0.76;
    }
};

struct hipComplex {
    float   r;
    float   i;
    hipComplex( float a, float b ) : r(a), i(b)  {}
    float magnitude2( void ) { return r * r + i * i; }
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
    float getx()
    {
	     return r;
    }
    float gety()
    {
	     return i;
    }
};

float julia( int x, int y ) {
    const float scale = 1;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
    //float res[3];
    //hipComplex c(-0.8, 0.156);
    hipComplex c(-0.82, 0.156);
    //hipComplex c(-0.65, 0.226);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 2)
            return 0;
    }
    LUT l(a.magnitude2());
    //res[0]= l.red();
    //res[1]= l.green();
    //res[2]= l.blue();
    return l.red()*255.0;
}

float julia2( int x, int y ) {
    const float scale = 1;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
    //float res[3];
    hipComplex c(-0.8, 0.149);
    //hipComplex c(-0.65, 0.226);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 2)
            return 0;
    }
    LUT l(a.magnitude2());
    //res[0]= l.red();
    //res[1]= l.green();
    //res[2]= l.blue();
    return l.green()*255.0;
}

float julia3( int x, int y ) {
    const float scale = 1;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
    //float res[3];
    hipComplex c(-0.8, 0.146);
    //hipComplex c(-0.65, 0.226);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 2)
            return 0;
    }
    LUT l(a.magnitude2());
    //res[0]= l.red();
    //res[1]= l.green();
    //res[2]= l.blue();
    return l.blue()*255.0;
}

void kernel( unsigned char *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;
	    //float f[3]= {julia( x, y )[0], julia( x, y )[1], julia( x, y )[2]};

            ptr[offset*4 + 0] = julia( x, y );
            ptr[offset*4 + 1] = julia2( x, y );
            ptr[offset*4 + 2] = julia3( x, y );
            ptr[offset*4 + 3] = 255;
        }
    }
 }

int main( void ) {
  CPUBitmap bitmap( DIM, DIM );
  unsigned char *ptr = bitmap.get_ptr();




  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  //record start event on the default stream
  hipEventRecord(start);//, 0);
  // Launch dot() kernel on GPU with N blocks

  kernel( ptr );

  //record stop event on the default stream
  hipEventRecord(stop);//, 0);
  //cout <<"GPU_result:"<< '\t'<<endl;
  //wait until the stop event completes
  hipEventSynchronize(stop);
  //calculate the elapsed time between two events
  float time;
  hipEventElapsedTime(&time, start, stop);


  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout <<""<<std::endl;
  std::cout <<"Kernel_time:           "<< time << '\t'<<std::endl;





  bitmap.display_and_exit();
}
