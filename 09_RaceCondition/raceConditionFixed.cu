#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void increment(int *d_x) {
  atomicAdd(d_x, 1); 
}

int main(){
  int x = 0, *d_x;

  hipMalloc((void**) &d_x, sizeof(int));
  hipMemcpy(d_x, &x, sizeof(int), hipMemcpyHostToDevice);

  increment<<<1000,1000>>>(d_x);

  hipMemcpy(&x, d_x, sizeof(int), hipMemcpyDeviceToHost);

  printf("x = %d\n", x);
  hipFree(d_x);
}

[13:05:30] jin6@titan1:~/CUDA/RaceCondition [69] nvcc  -Wno-deprecated-gpu-targets raceConditionFixed.cu
[13:06:28] jin6@titan1:~/CUDA/RaceCondition [70] ./a.out
x = 1000000



