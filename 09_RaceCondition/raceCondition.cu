#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void increment(int *d_x) {
  *d_x += 1;
}

int main(){
  int x = 0, *d_x;

  hipMalloc((void**) &d_x, sizeof(int));
  hipMemcpy(d_x, &x, sizeof(int), hipMemcpyHostToDevice);

  increment<<<1000,1000>>>(d_x);

  hipMemcpy(&x, d_x, sizeof(int), hipMemcpyDeviceToHost);

  printf("x = %d\n", x);
  hipFree(d_x);
}

