
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1024*1024 
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
    c[index] = a[index] + b[index];
}

void random_ints(int *x, int size)
{
  for (int i = 0; i < size; i++) {
    x[i] = rand() % 10;
  }
}

int main(void) {
  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *) malloc(size);
  random_ints(a, N);
  b = (int *) malloc(size);
  random_ints(b, N);
  c = (int *) malloc(size);
  
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N threads 
  add<<<(N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
  //add<<<(N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, 102500>>>(d_a, d_b, d_c, N);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) 
    printf("a[%d] = %d,  b[%d] = %d, c[%d] = %d\n", i, a[i], i, b[i], i, c[i]);

  // Cleanup
  hipFree(d_a); 
  hipFree(d_b);
  hipFree(d_c);
  
  return 0;
}

